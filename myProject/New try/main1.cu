#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>
#include <string.h>
#include "common.h"

#include "GPU_dp_uroll.h"



//to run as:
// ./main [random_or_not] [n_vols] [capacity] [random_seed] [blockDim.x]

int main(int argc, char **argv){

  int n_vols = 32;
  int* vols;
  int capacity = 10000;//12345678;

  //the first arguments tells if the sequence of volumes must be randomly generated (1)
  //or not (0)
  int generate_randomly_flag = 0;
  if(argc > 1){
    generate_randomly_flag = atoi(argv[1]);
  }

  //the second argument is the number of volumes. If 0, the default one is used.
  if(argc > 2){
    int _n_vols = atoi(argv[2]);
    if(_n_vols > 0){
      n_vols = _n_vols;
    }
  }
  vols = (int*) malloc(n_vols * sizeof(int));

  //the third argument is the total capacity. If 0, the default one is used.
  if(argc > 3){
    int _capacity = atoi(argv[3]);
    if(_capacity > 0){
      capacity = _capacity;
    }
  }

  //the fourth argument is the seed to be used in case of randomly generated volumes.
  //if 0, then the seed is randomized. Otherwise, the argument becomes the seed.

  if(generate_randomly_flag){
    int seed = 0;
    srand(time(0));
    if(argc > 4){
      seed = atoi(argv[4]);
      if(seed != 0){
        srand(seed);
      }
    }
    
    //"standard" values:
    //-lower = 50 ==> lower = capacity/200 
    //-upper = 500 ==> upper = capacity/20
    //-capacity = 10000;

    int lower = capacity/1000;
    int upper = capacity/10;
    for(int i = 0; i < n_vols; i++){
      vols[i] = (rand() % (upper - lower + 1) + lower);
      //printf("vols[%d] = %d\n", i, vols[i]);
    }

    //printf just to make sure the seed is correct during multiple runs
    printf("vols[%d] = %d\n", n_vols-1, vols[n_vols-1]);
  }else{
    for(int i = 0; i < n_vols; i++){
      vols[i] = 100*i;
    }
  }

  //check the volumes
  if(DEBUG_1){
    for(int i = 0; i < n_vols; i++){
      printf("vols[%d] = %d\n", i, vols[i]);
    }
  }
  
  
  
  
  
  
  

  
  double start, end;
  
  //first of all, following the algorithm, we have to agument the set of volumes with all the
  //powers of 2 smaller than capacity

  int* vols_agumented = (int*) malloc((n_vols + log2(capacity) + 1) * sizeof(int));
  for(int i = 0; i < n_vols; i++){
    vols_agumented[i] = vols[i];
  }

  for(int i = log2(capacity); i >= 0; i--){
    int add = pow(2, i);
    vols_agumented[n_vols + i] = add;
  }
  

  int res = capacity;
  int n_vols_agumented = n_vols + (int) log2(capacity) + 1;

  //----------------------------------------------------------------------------
  //-------------------------------GPU ALGORITHMS-------------------------------
  //----------------------------------------------------------------------------

  //if one wants to change the division between L1 cache and shared memory,
  //giving more shared memory for the smem approach or more L1 cache for the
  //kernels that don't use it
  CHECK(hipDeviceSetCacheConfig(hipFuncCachePreferL1));

  dim3 block(BLOCK_DIM_X);
  dim3 grid(((capacity + 1) + block.x - 1)/block.x);
  printf("block.x = %d, grid.x = %d\n", block.x, grid.x);

  //first, we need to declare host and device memory, and initialize it
  
  unsigned char* row_h = (unsigned char*) malloc((capacity + 1) * sizeof(unsigned char));
  if(row_h == NULL){
    printf("allocation failed!\n");
  }

  unsigned char *old_row_d, *new_row_d;
  CHECK(hipMalloc((unsigned char**)&old_row_d, (capacity + 1) * sizeof(unsigned char)));
  CHECK(hipMalloc((unsigned char**)&new_row_d, (capacity + 1) * sizeof(unsigned char)));
  CHECK(hipDeviceSynchronize());

  hipEvent_t eStart, eEnd;
  float msEvent;

  //--------------------------------GPU v2 uroll8--------------------------------
  
  dim3 grid8((grid.x + 8 - 1) / 8);
  start = seconds();
  res = capacity;
  n_vols_agumented = n_vols + (int) log2(capacity) + 1;

  CHECK(hipEventCreate(&eStart));
  CHECK(hipEventCreate(&eEnd));
  CHECK(hipEventRecord(eStart, 0));

  for(int i = 0; i<= (int) log2(capacity); i++){
    n_vols_agumented--;
    if(DP_v2_uroll8_GPU(vols_agumented, res, n_vols_agumented, row_h, old_row_d, new_row_d, grid8, block) == FALSE){
      res = res - vols_agumented[n_vols_agumented];
    }
  }

  CHECK(hipEventRecord(eEnd, 0));
  CHECK(hipEventSynchronize(eEnd));

  CHECK(hipEventElapsedTime(&msEvent, eStart, eEnd));

  end = seconds() - start;
  if(ON_MY_PC) end = (end / CLOCKS_PER_SEC) / 1000;
  if(ON_MY_PC) msEvent = msEvent / 1000;
  printf("DP v2 uroll8 GPU, res: %d, elapsed: %f, event time: %f\n", res, end * 1000, msEvent);

  CHECK(hipEventDestroy(eStart));
  CHECK(hipEventDestroy(eEnd));
  
  //--------------------------------GPU v2 uroll8 v2--------------------------------
  
  start = seconds();
  res = capacity;
  n_vols_agumented = n_vols + (int) log2(capacity) + 1;

  CHECK(hipEventCreate(&eStart));
  CHECK(hipEventCreate(&eEnd));
  CHECK(hipEventRecord(eStart, 0));

  for(int i = 0; i<= (int) log2(capacity); i++){
    n_vols_agumented--;
    if(DP_v2_uroll8_GPU_v2(vols_agumented, res, n_vols_agumented, row_h, old_row_d, new_row_d, grid8, block) == FALSE){
      res = res - vols_agumented[n_vols_agumented];
    }
  }

  CHECK(hipEventRecord(eEnd, 0));
  CHECK(hipEventSynchronize(eEnd));

  CHECK(hipEventElapsedTime(&msEvent, eStart, eEnd));

  end = seconds() - start;
  if(ON_MY_PC) end = (end / CLOCKS_PER_SEC) / 1000;
  if(ON_MY_PC) msEvent = msEvent / 1000;
  printf("DP v2 uroll8 v2 GPU, res: %d, elapsed: %f, event time: %f\n", res, end * 1000, msEvent);

  CHECK(hipEventDestroy(eStart));
  CHECK(hipEventDestroy(eEnd));

  




  //finally, release the memory

  

  free(row_h);

  CHECK(hipFree(old_row_d));
  CHECK(hipFree(new_row_d));
  CHECK(hipDeviceReset());








  





  free(vols_agumented);
  free(vols);


}


